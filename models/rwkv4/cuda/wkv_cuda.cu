#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#define MIN_VALUE (-1e38)

template <typename F>
__global__ void kernel_forward(const int B, const int T, const int C,
                               const F *__restrict__ const _w, const F *__restrict__ const _u, const F *__restrict__ const _k, const F *__restrict__ const _v,
                               F *__restrict__ const _y) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;  // idx in [0, B * C)
    const int _b = idx / C;  // batch index
    const int _c = idx % C;  // channel index
    const int _offset = _b * T * C + _c;  // offset

    F u = _u[_c];  // scalar
    F w = _w[_c];  // scalar
    const F *__restrict__ const k = _k + _offset;  // pointer from _k + _offset
    const F *__restrict__ const v = _v + _offset;  // pointer from _v + _offset
    F *__restrict__ const y = _y + _offset;  // pointer from _y + _offset

    F a_ = 0, b_ = 0, p = MIN_VALUE;  
    // a_ and b_ are running sums divided by exp(o) (to avoid overflows)
    for (int i = 0; i < T; i++) {
        const int t = i * C;

        // q = max(p_{t-1}, u + k_t)
        F q = max(p, u + k[t]);
        // wkv[t] = (exp(p_{t-1} - q) * a'_{t-1} + exp(u + k[t] - q) * v[t]) / (exp(p_{t-1} - q) * b'_{t-1} + exp(u + k[t] - q))
        y[t] = (exp(p - q) * a_ + exp(u + k[t] - q) * v[t]) / (exp(p - q) * b_ + exp(u + k[t] - q));

        F q_ = max(w + p, k[t]);  // q' = max(w + p_{t-1}, k_t)
        F A = exp(w + p - q_);  // A = exp(w + p_{t-1} - q')
        F B = exp(k[t] - q_);  // B = exp(k_t - q')
        a_ = A * a_ + B * v[t];  // a' = A * a' + B * v[t]
        b_ = A * b_ + B;  // b' = A * b' + B
        p = q_;  // p_{t-1} = q'
    }
}

template <typename F>
__global__ void kernel_backward(const int B, const int T, const int C,
                                const F *__restrict__ const _w, const F *__restrict__ const _u, const F *__restrict__ const _k, const F *__restrict__ const _v, const F *__restrict__ const _gy,
                                F *__restrict__ const _gw, F *__restrict__ const _gu, F *__restrict__ const _gk, F *__restrict__ const _gv) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;

    F u = _u[_c];
    F w = _w[_c];
    const F *__restrict__ const k = _k + _offset;
    const F *__restrict__ const v = _v + _offset;
    const F *__restrict__ const gy = _gy + _offset;

    F *__restrict__ const gk = _gk + _offset;
    F *__restrict__ const gv = _gv + _offset;

    F y[Tmax], z[Tmax], zexp[Tmax];

    F gw = 0, gu = 0;
    F p = 0, q = 0;
    F dpdw = 0, dqdw = 0;
    F o = MIN_VALUE;
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        F no = max(o, k[ii] + u);
        F A = exp(o - no);
        F B = exp(k[ii] + u - no);

        F num = A * p + B * v[ii];
        F iden = 1 / (A * q + B);

        y[i] = num * iden;
        z[i] = iden;
        zexp[i] = k[ii] + u - no;

        gw += gy[ii] * (dpdw - dqdw * y[i]) * iden * A;
        gu += gy[ii] * (v[ii] - y[i]) * B * iden;

        no = max(w + o, k[ii]);
        A = exp(w + o - no);
        B = exp(k[ii] - no);
        dpdw = A * (p + dpdw);
        dqdw = A * (q + dqdw);
        p = A * p + B * v[ii];
        q = A * q + B;
        o = no;
    }

    F gp = 0, gq = 0;
    o = MIN_VALUE;
    for (int i = T - 1; i >= 0; i--) {
        const int ii = i * C;
        F A = gy[ii] * z[i] * exp(zexp[i]);
        F B = exp(k[ii] + o);
        gk[ii] = A * (v[ii] - y[i]) + B * (gp * v[ii] + gq);
        gv[ii] = A + B * gp;

        F no = max(w + o, zexp[i] - k[ii] - u);
        A = exp(w + o - no);
        B = gy[ii] * z[i] * exp(zexp[i] - k[ii] - u - no);
        gp = A * gp + B;
        gq = A * gq - B * y[i];
        o = no;
    }

    // Multiply by w because the w -> -exp(w) preprocessing is halfway in the backwards pass, even though it's not in the forward pass
    const int _offsetBC = _b * C + _c;
    _gw[_offsetBC] += gw * _w[_c];
    _gu[_offsetBC] += gu;
}

void cuda_forward(int B, int T, int C, float *w, float *u, float *k, float *v, float *y) {
    dim3 threadsPerBlock(min(C, 32)); // requires --maxrregcount 60 for optimal performance
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_forward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, y);
}

void cuda_backward(int B, int T, int C, float *w, float *u, float *k, float *v, float *gy, float *gw, float *gu, float *gk, float *gv) {
    dim3 threadsPerBlock(min(C, 32)); // requires --maxrregcount 60 for optimal performance
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_backward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, gy, gw, gu, gk, gv);
}